
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


__global__ void scan(int size, int* input, int* output) {
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gidx < size) {
        output[gidx] = input[gidx];
    }
    __syncthreads(); 

    for (int stride = 1; stride < size; stride *= 2) {
        if (gidx >= stride && gidx < size) {
            output[gidx] = output[gidx] + output[gidx - stride];
        }
        __syncthreads(); 
    }
}


double get_clock() {
 struct timeval tv; int ok;
 ok = gettimeofday(&tv, (void *) 0);
 if (ok<0) { printf("gettimeofday error"); }
 return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main() {
  int size[] = {100, 1000, 10000, 100000, 1000000};
  for (int j = 0; j < 5; j++) {
     // allocate memory
     int* input;
     int* output;
     int* d_input;
     int* d_output;
  
     input = (int*)malloc(sizeof(int) * size[j]);
     output = (int*)malloc(sizeof(int) * size[j]);
     hipMalloc(&d_input, sizeof(int) * size[j]);
     hipMalloc(&d_output, sizeof(int) * size[j]);

     // initialize inputs
     for (int i = 0; i < size[j]; i++) {
     	 input[i] = 1;
     }
  
     hipMemcpy(d_input, input, sizeof(int) * size[j], hipMemcpyHostToDevice);
  
     // do the scan
     int thread = 128;
     double t0 = get_clock();
  
     scan<<<(size[j]+thread-1)/thread, thread>>>(size[j], d_input, d_output);
     hipDeviceSynchronize();
  
     double t1 = get_clock();
     printf("time for input size %d: %f s\n", size[j], t1-t0);
     hipMemcpy(output, d_output, sizeof(int) * size[j], hipMemcpyDeviceToHost);

     // free mem
     free(input);
     free(output);
     hipFree(d_input);
     hipFree(d_output);
  }

  return 0;
}
