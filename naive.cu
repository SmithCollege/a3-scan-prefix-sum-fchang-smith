
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 128

__global__ void scan(int size, int* input, int* output){
  int gidx = blockIdx.x*blockDim.x + threadIdx.x;
  output[gidx] = 0;
  for (int i  = 0; i < gidx+1; i++) {
      output[gidx] += input[i];
  }
}

double get_clock() {
 struct timeval tv; int ok;
 ok = gettimeofday(&tv, (void *) 0);
 if (ok<0) { printf("gettimeofday error"); }
 return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main() {
  // allocate memory
  int* input;
  int* output;
  int* d_input;
  int* d_output;
  
  input = (int*)malloc(sizeof(int) * SIZE);
  output = (int*)malloc(sizeof(int) * SIZE);
  hipMalloc(&d_input, sizeof(int) * SIZE);
  hipMalloc(&d_output, sizeof(int) * SIZE);

  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
  }
  
  hipMemcpy(d_input, input, sizeof(int) * SIZE, hipMemcpyHostToDevice);
  
  // do the scan
  int thread = 32;
  double t0 = get_clock();
  
  scan<<<(SIZE+SIZE-1)/thread, thread>>>(SIZE, d_input, d_output);
  hipDeviceSynchronize();
  
  double t1 = get_clock();
  printf("time for input size %d: %f s\n", SIZE, t1-t0);

  // check results
  hipMemcpy(output, d_output, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");

  // free mem
  free(input);
  free(output);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
